#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define LEN_LINHA 1024 // declara um número grande para poder caber tudo
#define LINHAS 100
#define COLUNAS 100

__global__ void mult_vetorxmatriz(int *matriz1, int *matriz2, int *resultado, int nColunas)
{
  int j = threadIdx.x;

  resultado[j] = 0;

  for (int i = 0; i < nColunas; i++)
  {
    resultado[j] += matriz1[i] * matriz2[i * COLUNAS + j];
  }
}

int main(int argc, char **argv)
{
  char *str;
  int linhas, colunas;
  char linha[LEN_LINHA]; // esse é o tamanho máximo de uma linha que o programa pode tratar
  dim3 grid_dims, block_dims;
  int *matriz, *vetor, *resultado;

  // Como não sabemos as dimensões máximas da matriz, vamos alocar uma quantidade
  // de memória "grande". Infelizmente é assim quando não sabemos qual vai ser o
  // tamanho da matriz

  hipMallocManaged(&vetor, sizeof(int) * COLUNAS);
  hipMallocManaged(&matriz, sizeof(int) * LINHAS * COLUNAS);
  hipMallocManaged(&resultado, sizeof(int) * LINHAS * COLUNAS);

  // vamos primeiro ler a linha e ver se o usuário digitou FIM
  while (fgets(linha, LEN_LINHA, stdin) != NULL && strcmp(linha, "FIM\n"))
  {
    // vamos separar a string de acordo com o separador
    str = strtok(linha, " "); // separa a string usando tokens
    colunas = 0;

    do
    {
      // converte para inteiro e armazena na matriz
      vetor[colunas] = strtol(str, NULL, 10);
      colunas++;

      // pega uma outra string. Caso seja um espaço vazio encerra o programa
    } while ((str = strtok(NULL, " ")) != NULL && strcmp(str, "\n"));

    linhas = 0;

    // lê uma linha da entrada padrão e testa para saber se não é uma
    // linha em branco, linha em branco encerra a leitura da matriz
    while (fgets(linha, LEN_LINHA, stdin) != NULL && strcmp(linha, "\n"))
    {
      colunas = 0;
      // vamos separar a string de acordo com o separador
      str = strtok(linha, " "); // separa a string usando tokens

      do
      {
        matriz[linhas * LINHAS + colunas] = strtol(str, NULL, 10);
        colunas++;

        // pega uma outra string. Caso seja um espaço vazio encerra o programa
      } while ((str = strtok(NULL, " ")) != NULL && strcmp(str, "\n"));
      linhas++; // vamos incrementar para no fim saber quantas linhas foram informadas
    }

    // === FIM do código para ler a matriz ===

    // Pronto. Quando chegar aqui já leu o número e a matriz.
    // As variáveis linhas e colunas tem as dimensões da matriz

    // É só fazer as contas

    grid_dims.x = 1;
    grid_dims.y = 1;
    grid_dims.z = 1;
    block_dims.x = colunas;
    block_dims.y = 1;
    block_dims.z = 1;

    mult_vetorxmatriz<<<grid_dims, block_dims>>>(vetor, matriz, resultado, colunas);
    hipDeviceSynchronize();

    // e depois imprimir
    for (int i = 0; i < 1; i++)
    {
      for (int j = 0; j < colunas; j++)
      {
        printf("%d ", resultado[i * COLUNAS + j]);
      }
      printf("\n");
    }
    printf("\n");
  }

  hipFree(vetor);
  hipFree(matriz);
  hipFree(resultado);

  return 0;
}