#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define LEN_LINHA   1024
#define LINHAS      100
#define COLUNAS     100

__global__ void multplicacaoEscalarMatriz(int linhas, int colunas, int escalar, int *matriz)
{
  for (int i = 1; i < linhas; i++)
  {
    for (int j = 0; j < colunas; j++)
    {
      int multiplicacao = matriz[i*COLUNAS+j] * escalar;
      matriz[i*COLUNAS+j] = multiplicacao;
    }
  }
  return;
}

int main()
{
  char *str;
  int *matriz; // host
  char linha[LEN_LINHA];
  int linhas, colunas, escalar;

  hipMallocManaged(&matriz, sizeof(int)*LINHAS*COLUNAS); // device

  while(fgets(linha, LEN_LINHA, stdin) != NULL && strcmp(linha, "FIM\n")) {
    sscanf(linha, "%d", &escalar);

    linhas = 0;

    while(fgets(linha, LEN_LINHA, stdin) != NULL && strcmp(linha, "\n")) {
      colunas = 0;

      str = strtok(linha, " "); // Separa a string usando tokens    ['1', '2', '3']
      do {
        matriz[linhas*LINHAS+colunas] = strtol(str, NULL, 10);
        colunas++;
      }while((str=strtok(NULL, " "))  != NULL && strcmp(str, "\n"));
      linhas++;
    }

    multplicacaoEscalarMatriz<<<1, 1>>>(linhas, colunas, escalar, matriz);
    hipDeviceSynchronize();

    for(int i=0; i<linhas; i++){
      for(int j=0; j<colunas; j++){
        printf("%2d ", matriz[i*COLUNAS+j]);
      }
      printf("\n");
    }
    printf("\n");

  }
  
  hipFree(matriz);

  return 0;
}