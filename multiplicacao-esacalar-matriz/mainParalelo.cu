#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define LEN_LINHA 1024
#define LINHAS 100
#define COLUNAS 100

__global__ void multplicacaoEscalarMatriz(int linhas, int colunas, int escalar, int *matriz)
{
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  int col = blockDim.y  * blockIdx.y + threadIdx.y;

  printf("%d\n", row);
  printf("%d\n", col);
  for (int i = 0; i < linhas; i++)
  {
    for (int j = 0; j < colunas; j++)
    {
      int multiplicacao = matriz[i*COLUNAS+j] * escalar;
      matriz[i*COLUNAS+j] = multiplicacao;
    }
  }
  return;
}

int main()
{
  char *str;
  int *matriz;
  char linha[LEN_LINHA];
  int linhas, colunas, escalar, threadPerBlock;

  hipMallocManaged(&matriz, sizeof(int) * LINHAS * COLUNAS);

  while (fgets(linha, LEN_LINHA, stdin) != NULL && strcmp(linha, "FIM\n"))
  {
    sscanf(linha, "%d", &escalar);

    linhas = 0;

    while (fgets(linha, LEN_LINHA, stdin) != NULL && strcmp(linha, "\n"))
    {
      colunas = 0;

      str = strtok(linha, " "); // Separa a string usando tokens
      do
      {
        matriz[linhas * LINHAS + colunas] = strtol(str, NULL, 10);
        colunas++;
      } while ((str = strtok(NULL, " ")) != NULL && strcmp(str, "\n"));
      linhas++;
    }

    threadPerBlock = linhas*colunas;

    multplicacaoEscalarMatriz<<<1, threadPerBlock>>>(linhas, colunas, escalar, matriz);
    hipDeviceSynchronize();

    for (int i = 0; i < linhas; i++)
    {
      for (int j = 0; j < colunas; j++)
      {
        printf("%2d ", matriz[i * COLUNAS + j]);
      }
      printf("\n");
    }
    printf("\n");
  }

  hipFree(matriz);

  return 0;
}