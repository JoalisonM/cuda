#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

int arrayPosition = 0;

__global__ void multplicacaoEscalarMatriz(int linhas, int colunas, int escalar, int *matriz, int *inputs){
  for (int i = 0; i < linhas; i++)
  {
    for (int j = 0; j < colunas; j++)
    {
      int multiplicacao = inputs[i * colunas + j] * escalar;
      matriz[i*colunas+j] = multiplicacao;
    }
  }
  return;
}

void readLine(char *a)
{
  switch (scanf("%9999999[^\n]", a))
  {
  case 1:
    break;
  case 0:
    a[0] = 0;
    break;
  case EOF:
    return;
  }
  int ch;
  while ((ch = fgetc(stdin)) != '\n' && ch != EOF)
    ;
}

int stringToInt(char *string, int size)
{
  int ehNegativo = 0;

  if (string[0] == '-')
    ehNegativo = 1;

  int answer = 0;
  for (int i = ehNegativo; i < size; i++)
  {
    if (string[i] > '9' || string[i] < '0')
      break;
    answer *= 10;
    answer += string[i] - '0';
  }
  return (ehNegativo == 1 ? -answer : answer);
}

int getItems(int *inputs, char *string, int sizeString)
{
  int colunas = 0;
  char number[10];
  int position = 0;

  for (int i = 0; i < sizeString; i++)
  {
    if (string[i] != ' ')
    {
      number[position] = string[i];
      position++;
    }
    else
    {
      int n = stringToInt(number, position);
      inputs[arrayPosition] = n;
      (arrayPosition)++;
      position = 0;
    }
  }

  for (int i = 0; i < sizeString; i++)
    if (string[i] == ' ')
      colunas++;

  return colunas;
}

int main()
{

  while (1)
  {
    char initialNumber[1000000];
    readLine(initialNumber);
    if (strcmp(initialNumber, "FIM") == 0)
      break;

    int escalar = stringToInt(initialNumber, 18);
    int linhas = 0, colunas;
    int *matriz, *inputs;
    arrayPosition = 0;
  
    hipMallocManaged(&inputs, sizeof(int)*1000000);

    while (1)
    {
      char line[1000000];
      readLine(line);
      if (strcmp(line, "") == 0)
        break;
      colunas = getItems(inputs, line, strlen(line));
      linhas++;
    }

    hipMallocManaged(&matriz, sizeof(int)*linhas*colunas);
  
    multplicacaoEscalarMatriz<<<1, 1>>>(linhas, colunas, escalar, matriz, inputs);
    hipDeviceSynchronize();
    
    for (int i = 0; i < linhas; i++)
    {
      for (int j = 0; j < colunas; j++)
      {
        printf("%2d ", matriz[i*colunas+j]);
      }
      printf("\n");
    }
    printf("\n");

    hipFree(inputs);
    hipFree(matriz);
  }

  return 0;
}