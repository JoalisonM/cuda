#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

int inputs[1000000];
int arrayPosition = 0;

// __global__ void multplicacaoEscalarMatriz(int escalar, int *matriz)


void readLine(char *a)
{
  switch (scanf("%9999999[^\n]", a))
  {
  case 1:
    break;
  case 0:
    a[0] = 0;
    break;
  case EOF:
    return;
  }
  int ch;
  while ((ch = fgetc(stdin)) != '\n' && ch != EOF)
    ;
}

int stringToInt(char *string, int size)
{
  int ehNegativo = 0;

  if (string[0] == '-')
    ehNegativo = 1;

  int answer = 0;
  for (int i = ehNegativo; i < size; i++)
  {
    if (string[i] > '9' || string[i] < '0')
      break;
    answer *= 10;
    answer += string[i] - '0';
  }
  return (ehNegativo == 1 ? -answer : answer);
}

int getItems(char *string, int sizeString)
{
  int colunas = 1;
  char number[10];
  int position = 0;

  for (int i = 0; i < sizeString; i++)
  {
    if (string[i] != ' ')
    {
      number[position] = string[i];
      position++;
    }
    else
    {
      int n = stringToInt(number, position);
      inputs[arrayPosition] = n;
      (arrayPosition)++;
      position = 0;
    }
  }

  int n = stringToInt(number, position);
  inputs[arrayPosition] = n;
  (arrayPosition)++;
  position = 0;

  for (int i = 0; i < sizeString; i++)
    if (string[i] == ' ')
      colunas++;

  return colunas;
}

int main()
{
  int *matriz;
  int linhas = 0, colunas;
  while (1)
  {
    char initialNumber[1000000];
    readLine(initialNumber);
    if (strcmp(initialNumber, "FIM") == 0)
      break;

    int escalar = stringToInt(initialNumber, 18);
    arrayPosition = 0;

    while (1)
    {
      char line[1000000];
      readLine(line);
      if (strcmp(line, "") == 0)
        break;
      colunas = getItems(line, strlen(line));
      linhas++;
    }

    hipMallocManaged(&matriz, sizeof(int)*linhas*colunas);

    for (int i = 0; i < linhas; i++)
    {
      for (int j = 0; j < colunas; j++)
      {
        *matriz = inputs[i * colunas + j] * escalar;
        printf("%2d ", *matriz);
      }
      printf("\n");
    }
    printf("\n");
    hipFree(matriz);
  }

  return 0;
}