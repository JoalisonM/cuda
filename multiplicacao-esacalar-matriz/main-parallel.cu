#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define LEN_LINHA 1024 // declara um número grande para poder caber tudo
#define LINHAS 100
#define COLUNAS 100

__global__ void mult_EscalarXMatriz(int escalar, int *matriz, int nlinhas, int ncolunas)
{
  // threadIdx.y e threadIdx.x contém a linha e a coluna respectivamente deste thread no grid
  int i = threadIdx.y; // linha
  int j = threadIdx.x; // coluna

  matriz[i * COLUNAS + j] = escalar * matriz[i * COLUNAS + j];
}

int main()
{
  char *str;
  int *matriz;
  char linha[LEN_LINHA]; // esse é o tamanho máximo de uma linha que o programa pode tratar
  dim3 grid_dims, block_dims;
  int linhas, colunas, numero;

  // Como não sabemos as dimensões máximas da matriz, vamos alocar uma quantidade
  // de memória "grande". Infelizmente é assim quando não sabemos qual vai ser o
  // tamanho da matriz
  hipMallocManaged(&matriz, sizeof(int) * LINHAS * COLUNAS);

  while (fgets(linha, LEN_LINHA, stdin) != NULL && strcmp(linha, "FIM\n"))
  {
    // Agora que leu e sabemos  que é diferente de FIM vamos ler o número a ser multiplicado
    sscanf(linha, "%d", &numero);

    // === Agora vamos ler a matriz ===
    linhas = 0;

    // Lê uma linha da entrada padrão e testar para saber se não é uma
    // linha em branco. Se for encerra a leitura da matriz
    while (fgets(linha, LEN_LINHA, stdin) != NULL && strcmp(linha, "\n"))
    {
      colunas = 0;

      // Vamos separar a string de acordo com o separador
      str = strtok(linha, " "); // Separa a string usando tokens
      do
      {
        // Converte para inteiro e armazena na matriz
        matriz[linhas * LINHAS + colunas] = strtol(str, NULL, 10);
        colunas++;

        // Pega uma outra string. Caso seja um espaço vazio encerra o programa
      } while ((str = strtok(NULL, " ")) != NULL && strcmp(str, "\n"));
      linhas++; // Vamos incrementar para no fim saber quantas linhas foram informadas
    }

    // === Fim do código para ler a matriz ===

    // Pronto. Quando chegar aqui já leu o número e a matriz.
    // As variáveis linhas e colunas tem as dimensões da matriz.

    grid_dims.x = 1;
    grid_dims.y = 1;
    grid_dims.z = 1;
    block_dims.x = colunas;
    block_dims.y = linhas;
    block_dims.z = 1;

    // Cria os threads
    mult_EscalarXMatriz<<<grid_dims, block_dims>>>(numero, matriz, linhas, colunas);

    // Aguarda a conclusão de todos eles
    hipDeviceSynchronize();

    // Imprimi a matriz
    for (int i = 0; i < linhas; i++)
    {
      for (int j = 0; j < colunas; j++)
      {
        printf("%d ", matriz[i * COLUNAS + j]);
      }
      printf("\n");
    }
    printf("\n");
  }

  hipFree(matriz);

  return 0;
}