#include <stdio.h>
#include <hip/hip_runtime.h>

int main(int argc, char **argv)
{
  int *matriz;
  int linhas, colunas;

  while ((scanf("%d %d", &linhas, &colunas) == 2) && (linhas > 0 && colunas > 0))
  {
    hipMallocManaged(&matriz, sizeof(int)*linhas*colunas);

    for (int i = 0; i < linhas; i++)
    {
      for (int j = 0; j < colunas; j++)
      {
        scanf("%d", &matriz[i*colunas+j]);
      }
    }

    for (int i = 0; i < linhas; i++)
    {
      for (int j = 0; j < colunas; j++)
      {
        printf("%2d", matriz[i*colunas+j]);
      }
      printf("\n");
    }
    printf("\n");
    hipFree(matriz);
  }

  return 0;
}