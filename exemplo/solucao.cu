#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define LEN_LINHA   1024    // Declara um número grande para poder caber tudo
#define LINHAS      100
#define COLUNAS     100

int main(int argc, char **argv){
    char *str;
    int *matriz;
    int linhas, colunas, numero;
    char linha[LEN_LINHA]; // Esse é o tamanho máximo de uma linha que o programa pode tratar

    // Como não sabemos as dimensões máximas da matriz, vamos alocar uma quantiade
    // de memória "grande". Infelizmente é assim quando não sabemos qual vai ser o 
    // tamanho da matriz
    hipMallocManaged(&matriz, sizeof(int)*LINHAS*COLUNAS);

    // Vamos primeiro ler a linha e ver se o usuário digitou FIM
    while(fgets(linha, LEN_LINHA, stdin) != NULL && strcmp(linha, "FIM\n")){
        // Agora que leu e sabemos que é diferente de FIM vamos ler o número  a ser
        // multiplicado
        sscanf(linha, "%d", &numero);
        
        // ============== Agora vamos ler a matriz ===================================
        linhas = 0;
        // Le uma linha da entrada padrão e testa para saber se não é uma linha em branco
        // linha em branco encerra a leitura da matriz
        while(fgets(linha, LEN_LINHA, stdin) != NULL && strcmp(linha, "\n")){
            colunas = 0;
            // Vamos separar a string de acordo com o separador
            str = strtok(linha," "); // Separa a string usando tokens
            do{ 
                // Converte para inteiro e armazena na matriz
                matriz[linhas*LINHAS+colunas] = strtol(str, NULL, 10);
                colunas++;

                // Pega uma outra string. Caso seja um espaço vazio encerra o programa
            }while((str=strtok(NULL, " ")) != NULL && strcmp(str, "\n")); 
            linhas++; // Vamos incrementar para no fim saber quantas linhas foram informadas
        }
        // ============= Fim do código para ler a matriz ============================

        // Pronto. Quando chegar aqui já leu o número e a matriz. 
        // As variáveis linhas e colunas tem as dimensões da matriz

        // Eh só fazer as contas

        // E depois imprimir
        for(int i=0; i<linhas; i++){
            for(int j=0; j<colunas; j++){
                printf("%d ", matriz[i*LINHAS+j]);
            }
            printf("\n");
        }
    }

    hipFree(matriz);

    return 0;
}