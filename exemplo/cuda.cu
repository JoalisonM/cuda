#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define f(x) 1 + sin(x)

__global__ void Dev_trap(
    const float a, // in
    const float h, // in
    const int   n, // in
    float *trap_p  // in/out
){
    int my_id = blockDim.x * blockIdx.x + threadIdx.x;

    if(0<my_id && my_id<n){
        float my_x = a + my_id*h;
        float my_trap = f(my_x);
        atomicAdd(trap_p, my_trap);
    }
}

float trap_wrapper(
    const float a, // Entrada. Limite inferior
    const float b, // Entrada. Limite superior
    const int   blk_ct, // Entrada. Tamanho do grid de processadores
    const int   th_per_blk // Entrada. Threads por por bloco de processadores
    ){
    float *trap_p;
    // Número de divisões (trapézios no intervalo)
    int n = blk_ct*th_per_blk;
    float h = (b-a)/n;
    float result = (1.0/2.0)*(f(a) + f(b));

    hipMallocManaged(&trap_p, sizeof(float));

    *trap_p = 0;

    Dev_trap<<<blk_ct, th_per_blk>>>(a, h, n, trap_p);
    hipDeviceSynchronize();

    result = h*(result + (*trap_p));

    hipFree(trap_p);

    return result;
}

int main(int argc, char ** argv){
    float a, b, result;
    int blk_ct, th_per_blk;

    printf("Informe o limite inferior e superior do intervalo: ");
    if(scanf("%f %f", &a, &b)!=2) return 1;
    printf("Informe o número de blocos e threads por bloco: ");
    if(scanf("%d %d", &blk_ct, &th_per_blk)!=2) return 1;

    result = trap_wrapper(a, b, blk_ct, th_per_blk);

    printf("A aproximação vale: %.2f\n", result);

    return 0;
}