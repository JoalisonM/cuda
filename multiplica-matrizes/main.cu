#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_LINHAS 10
#define MAX_COLUNAS 10

__global__ void mult_matrizxmatriz(long int *matriz1, long int *matriz2, long int *resultado, int nlinhas1, int ncolunas1, int ncolunas2)
{
  int i = threadIdx.y; // linhas da matriz 1
  int j = threadIdx.x; // colunas da matriz 2

  resultado[i * ncolunas2 + j] = 0;

  for (int k = 0; k < ncolunas1; k++)
  {
    resultado[i * ncolunas2 + j] += matriz1[i * ncolunas1 + k] * matriz2[k * ncolunas2 + j];
  }
}

int main(int argc, char **argv)
{
  dim3 grid_dims, block_dims;
  long int *matriz1, *matriz2, *resultado;
  int nlinhas1, ncolunas1, nlinhas2, ncolunas2;

  // vou colocar as matrizes nas dimensões máximas mas caso elas fossem
  // grandes o ideal é alocar segundo a demanda.

  hipMallocManaged(&matriz1, sizeof(int) * MAX_LINHAS * MAX_COLUNAS);
  hipMallocManaged(&matriz2, sizeof(int) * MAX_LINHAS * MAX_COLUNAS);
  hipMallocManaged(&resultado, sizeof(int) * MAX_LINHAS * MAX_COLUNAS);

  while ((scanf("%d %d", &nlinhas1, &ncolunas1) == 2) && nlinhas1 > 0 && ncolunas1 > 0)
  {
    for (int i = 0; i < nlinhas1; i++)
    {
      for (int j = 0; j < ncolunas1; j++)
      {
        if (scanf("%ld", &matriz1[i * ncolunas1 + j]) != 1)
          exit(1);
      }
    }

    nlinhas2 = ncolunas1;

    if (scanf("%d", &ncolunas2) != 1)
      exit(1);

    for (int i = 0; i < nlinhas2; i++)
    {
      for (int j = 0; j < ncolunas2; j++)
      {
        if (scanf("%ld", &matriz2[i * ncolunas2 + j]) != 1)
          exit(1);
      }
    }

    grid_dims.x = 1;
    grid_dims.y = 1;
    grid_dims.z = 1;
    block_dims.x = ncolunas2;
    block_dims.y = nlinhas1;
    block_dims.z = 1;

    mult_matrizxmatriz<<<grid_dims, block_dims>>>(matriz1, matriz2, resultado, nlinhas1, ncolunas1, ncolunas2);
    hipDeviceSynchronize();

    for (int i = 0; i < nlinhas1; i++)
    {
      for (int j = 0; j < ncolunas2; j++)
      {
        printf("%2ld ", resultado[i * ncolunas2 + j]);
      }
      printf("\n");
    }
    printf("\n");
  }

  hipFree(matriz1);
  hipFree(matriz2);
  hipFree(resultado);

  return 0;
}